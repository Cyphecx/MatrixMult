#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrix.h"

 __global__ void multiplyMatrix(Datatype* matrix1, Datatype* matrix2, Datatype* matrix3, int size, int quantity, int offset){
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_index < size*quantity){
        return;
    }
    // index = y * size + x;
    int x = thread_index % size;
    int y = thread_index / size;
    double sum = 0;
    for(int k = 0; k < size; ++k){
        sum += matrix1[y*size + k] * matrix2[k*size + x];
    }
    matrix3[thread_index] = sum;
 }



extern "C" void initCuda(Datatype* matrix1, Datatype* matrix2, Datatype* matrix3, int size, int rank, int quantity) {
    //rank should be decremented by one
    int cudaDeviceCount = 0;
    int cE;
    //Error checking
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
               cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
               rank, (rank % cudaDeviceCount), cE);
        exit(-1);
    }

    hipMallocManaged(&matrix1, size*quantity*sizeof(Datatype));
    hipMallocManaged(&matrix2, size*size*sizeof(Datatype));
    hipMallocManaged(&matrix3, size*size*sizeof(Datatype));

}

extern "C" void multiply(Datatype* matrix1, Datatype* matrix2, Datatype* matrix3, int size, int quantity, int offset) {

    int block_count = (size*quantity) / 32 + 1;
    multiplyMatrix<<<block_count,32>>>(matrix1, matrix2, matrix3, size, quantity, offset);
}
